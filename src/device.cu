#include "hip/hip_runtime.h"
__device__ __forceinline__
int search(
    const int * __restrict__ vec,
    int lower,
    int upper,
    const int search
) {
    int middle;
    while (lower < upper) {
        middle = lower + ((upper - lower) >> 1);
        if (vec[middle] < search) {
            lower = middle + 1;
        } else {
            upper = middle;
        }
    }
    return lower;
}

__device__ __forceinline__
int2 search_2key(
    const int * __restrict__ arr,
    int left,
    int right,
    const int small_key,
    const int large_key
) {
    int middle;
    int2 out = make_int2(-1, -1);
    while (left <= right) {
        middle = left + ((right - left) >> 1);
        if (arr[middle] < small_key)  {
            left = middle+1;
        } else if (arr[middle] == small_key) {
            out.x = middle;
            out.y = search(arr, middle+1, right, large_key);
            break;
        } else if (arr[middle] > small_key && arr[middle] < large_key) {
            if (left <= middle) {
                out.x = search(arr, left, middle, small_key);
            } else {
                out.x = middle;
            }

            if (middle+1 <= right) {
                out.y = search(arr, middle+1, right+1, large_key);
            } else {
                out.y = middle+1;
            }
            break;
        } else if (arr[middle] == large_key) {
            out.y = middle;
            out.x = search(arr, left, middle-1, small_key);
            break;
        } else if (arr[middle] > large_key) {
            right = middle-1;
        }
    }
    return out;
}

__global__
void init(
    int * __restrict__ v1,
    int * __restrict__ v2,
    int numels
) {
    int i = getId();
    if (i >= numels) return;

    v1[i] = 2*i;
    v2[i] = 2*i + 1;
}

__global__
void merge(
    const int2 * __restrict__ v1,
    const int2 * __restrict__ v2,
    int * __restrict__ vmerge,
    int numels
) {
    // Iterativo
    /*
    for (int i = 0; i < numels; ++i) {
        i2 = search(v2, v1[i])
        vmerge[i+i2] = v1[i];
    }

    // tutti gli indici rimanenti [k] = v2[j]
    for (int i = 0; i < 2*numels; ++i) {
        if (vmerge[i] == -1) vmerge[i] = v2[i];
    }
    */
    int i = getId();
    if (i >= numels/2) return;

    int2 el1 = v1[i];
    int2 index_el1_in_v2 = search_2key((int*)v2, 0, numels-1, el1.x, el1.y); // 2*i;
    vmerge[(i*2) + (index_el1_in_v2.x + 1)] = el1.x;
    vmerge[(i*2) + (index_el1_in_v2.y + 1)] = el1.y;

    int2 el2 = v2[i];
    int2 index_el2_in_v1 = search_2key((int*)v1, 0, numels-1, el2.x, el2.y); // i1+1;
    vmerge[(i*2) + (index_el2_in_v1.x + 1)] = el2.x;
    vmerge[(i*2) + (index_el2_in_v1.y + 1)] = el2.y;

    // sincronizza e aggiungi i mancanti
    // __syncthreads();
    // if (i == 0) {
    // 	for (int j = 0, j2 = 0; (j < 2*numels) && (j2 < numels); ++j) {
    // 		if (vmerge[j] == -1) {
    // 			vmerge[j] = v2[j2];
    // 			++j2;
    // 		}
    // 	}
    // }

}
