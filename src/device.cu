#include "hip/hip_runtime.h"
__device__ __forceinline__
int search(
    const int * __restrict__ vec,
    const int search,
    const int length
) {
    int lower = 0;
    int upper = length;
    int middle;
    while (lower < upper) {
        middle = (lower + upper) >> 1;
        if (vec[middle] < search) {
            lower = middle + 1;
        } else {
            upper = middle;
        }
    }
    return lower;
}

__global__
void init(
    int * __restrict__ v1,
    int * __restrict__ v2,
    int numels
) {
    int i = getId();
    if (i >= numels) return;

    v1[i] = 2*i;
    v2[i] = 2*i + 1;
}

__global__
void merge(
    const int * __restrict__ v1,
    const int * __restrict__ v2,
    int * __restrict__ vmerge,
    int numels
) {
    // Iterativo
    /*
    for (int i = 0; i < numels; ++i) {
        i2 = search(v2, v1[i])
        vmerge[i+i2] = v1[i];
    }

    // tutti gli indici rimanenti [k] = v2[j]
    for (int i = 0; i < 2*numels; ++i) {
        if (vmerge[i] == -1) vmerge[i] = v2[i];
    }
    */
    int i = getId();
    if (i >= numels) return;

    int el1 = v1[i];
    int el2 = v2[i];

    int index_el1_in_v2 = search(v2, el1, numels); // 2*i;
    int index_el2_in_v1 = search(v1, el2, numels); // i1+1;

    vmerge[i+index_el1_in_v2] = el1;
    vmerge[i+index_el2_in_v1] = el2;

    // sincronizza e aggiungi i mancanti
    // __syncthreads();
    // if (i == 0) {
    // 	for (int j = 0, j2 = 0; (j < 2*numels) && (j2 < numels); ++j) {
    // 		if (vmerge[j] == -1) {
    // 			vmerge[j] = v2[j2];
    // 			++j2;
    // 		}
    // 	}
    // }

}
